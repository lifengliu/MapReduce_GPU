
#include <hip/hip_runtime.h>
#include <dirent.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <malloc.h>

#define MAP_COUNT __device__ void mapCount(char*key,char*value,size_t key_size, size_t value_size,int*key_im_size,int*value_im_size,int*map_im_num,int threadID)
#define EMIT_IM_COUNT(im_key_size,im_value_size) emitMapCount(im_key_size,im_value_size,word_num,key_im_size,value_im_size,map_im_num,threadID)

/*extern */MAP_COUNT;


typedef struct MapFileList {
	char* filename;
	struct MapFileList* next;
}MapFileList;

typedef enum InputFormat{TextInputFormat,KeyValueInputFormat,SequenceFileInputFormat} input_format;

typedef struct Index{
	int key_offset;
	int key_size;
	int value_offset;
	int value_size;
}Index;

typedef struct MapReduceSpec{
	MapFileList* map_file_list;
	char* map_input_keys;
	char* map_input_values;
	Index* map_input_index;
	int* map_im_key_size;
	int* map_im_value_size;
	int* map_im_num;
	char* im_keys;
	int* im_values;
	Index* im_index;
	int map_input_num;
	int map_block_num;
	int map_thread_num;
	input_format map_input_format;
}MapReduceSpec;

char* d_map_input_keys;
char* d_map_input_values;
Index* d_map_input_index;

void init_map_file_list(MapFileList* list){
	list->filename=NULL;
	list->next=NULL;
}

void free_map_file_list(MapFileList* list){
	MapFileList* del;
	MapFileList* tmp;
	del=list;
	tmp=list->next;
	while(tmp){
		if(del->filename!=NULL)
			free(del->filename);
		free(del);
		del=tmp;
		tmp=tmp->next;
	}
	if(del->filename!=NULL)
		free(del->filename);
	free(del);	
}

void init_mapreduce_spec(MapReduceSpec* spec){
	spec->map_file_list=NULL;
	spec->map_input_keys=NULL;
	spec->map_input_values=NULL;
	spec->map_input_index=NULL;
	spec->map_im_key_size=NULL;
	spec->map_im_value_size=NULL;
	spec->map_im_num=NULL;
	spec->im_keys=NULL;
	spec->im_values=NULL;
	spec->im_index=NULL;
	spec->map_input_num=0;
	spec->map_block_num=0;
	spec->map_thread_num=512;
	spec->map_input_format=TextInputFormat;
}

void free_spec(MapReduceSpec* spec){
	free_map_file_list(spec->map_file_list);
	free(spec->map_input_keys);
	free(spec->map_input_values);
	free(spec->map_input_index);
	free(spec->map_im_key_size);
	free(spec->map_im_value_size);
	free(spec->map_im_num);
	free(spec->im_keys);
	free(spec->im_values);
	free(spec->im_index);
	free(spec);
}

char *my_strncpy(char *dest, const char *src, size_t n)
{
    size_t i;

   for (i = 0; i < n && src[i] != '\0'; i++)
        dest[i] = src[i];
    for ( ; i < n; i++)
        dest[i] = '\0';

   return dest;
}

void map_input_split(MapReduceSpec* spec){
	MapFileList* file_list_entry;
	size_t buffer_size=(size_t)256*1024*1024;
	size_t buffer_used=0;

	FILE* pFile;
	file_list_entry=spec->map_file_list;	
	
	size_t file_size;
	size_t key_array_size;
	size_t value_array_size;
	size_t index_array_size;

	if(spec->map_input_format==TextInputFormat){
		file_size=key_array_size=value_array_size=index_array_size=0;

		while(file_list_entry->filename!=NULL){
			pFile=fopen(file_list_entry->filename,"rb");
			if (pFile==NULL) {fputs ("File error\n",stderr); exit (1);}
			fseek (pFile , 0 , SEEK_END);
			file_size = ftell (pFile);
			rewind (pFile);
			if(buffer_used+file_size<=buffer_size){
				ssize_t result=0;
				while (result!= -1) {					
					size_t value_size = 0;
					size_t key_size=0;
					char* temp_key=NULL;
					char* temp_value=NULL;

					temp_key=(char*)malloc(10);
					sprintf(temp_key,"%d",(int)ftell(pFile));
					key_size=strlen(temp_key)+1;                      //get the new key's size

					spec->map_input_keys=(char*)realloc(spec->map_input_keys,key_array_size+key_size);             //reallocate key_array, so that it can contain new keys
					my_strncpy((spec->map_input_keys)+key_array_size,temp_key,key_size);
					result=getline(&(temp_value), &value_size, pFile);
					value_size=strlen(temp_value)+1;
					spec->map_input_values=(char*)realloc(spec->map_input_values,value_array_size+value_size);           //reallocate value_size, so that it can contain new values
					strcpy((char*)(spec->map_input_values+value_array_size),temp_value);
					spec->map_input_index=(Index*)realloc(spec->map_input_index,(index_array_size+1)*sizeof(Index));            //reallocate index array, so that it can contain new <key,value> information

					spec->map_input_index[index_array_size].key_offset=key_array_size;
					spec->map_input_index[index_array_size].key_size=key_size;					
					spec->map_input_index[index_array_size].value_offset=value_array_size;
					spec->map_input_index[index_array_size].value_size=value_size;

					key_array_size+=key_size;
					value_array_size+=value_size;
					index_array_size++;
					free(temp_key); free(temp_value);
				}
				buffer_used=buffer_used+file_size;
			}
			else
				printf("Buffer full!!\n");
			file_list_entry=file_list_entry->next;
			fclose(pFile);
		}
		spec->map_input_num=index_array_size;
//		printf("Map Input entry number: %i, %u, %u, %u\n",spec->map_input_num,key_array_size,value_array_size,index_array_size*sizeof(Index));
		printf("Map Input entry number: %i\n",spec->map_input_num);
	}	

}

__device__ bool isChar(char c){
	if(((c<='z')&&(c>='a'))||((c<='Z')&&(c>='A')))
		return true;
	else
		return false;
}

__device__ void emitMapCount(int key_size, int value_size,int word_num,int*key_im_size_array,int*value_im_size_array,int*map_im_num,int threadID){
	*(key_im_size_array+threadID)=key_size;
	*(value_im_size_array+threadID)=value_size;
	*(map_im_num+threadID)=word_num;
}

__global__ void map_count_warp(char*keys,char*values,Index*index,int*map_im_key_size,int*map_im_value_size,int*map_im_num,int input_num){
	int i=blockDim.x*blockIdx.x+threadIdx.x;
	if(i<input_num){
		mapCount((keys+((index+i)->key_offset)),(values+((index+i)->value_offset)),(index+i)->key_size,(index+i)->value_size,map_im_key_size,map_im_value_size,map_im_num,i);
	}
}

void map_count_phase(MapReduceSpec* spec){
	// char* d_map_input_keys;
	// char* d_map_input_values;
	// Index* d_map_input_index;
	int* d_map_im_key_size;
	int* d_map_im_value_size;
	int* d_map_im_num;
	size_t map_im_size=(spec->map_input_num)*sizeof(int);
	spec->map_im_key_size=(int*)malloc(map_im_size);
	spec->map_im_value_size=(int*)malloc(map_im_size);
	spec->map_im_num=(int*)malloc(map_im_size);

	size_t keys_size=malloc_usable_size(spec->map_input_keys);
	size_t values_size=malloc_usable_size(spec->map_input_values);
	size_t index_size=malloc_usable_size(spec->map_input_index);
	//printf("%u,%u,%u\n",malloc_usable_size(spec->map_input_keys),malloc_usable_size(spec->map_input_values),malloc_usable_size(spec->map_input_index));
	hipMalloc(&d_map_input_keys,keys_size);
	hipMalloc(&d_map_input_values,values_size);
	hipMalloc(&d_map_input_index,index_size);
	hipMalloc(&d_map_im_key_size,map_im_size);
	hipMalloc(&d_map_im_value_size,map_im_size);
	hipMalloc(&d_map_im_num,map_im_size);
	hipMemcpy(d_map_input_keys,spec->map_input_keys,keys_size,hipMemcpyHostToDevice);
	hipMemcpy(d_map_input_values,spec->map_input_values,values_size,hipMemcpyHostToDevice);
	hipMemcpy(d_map_input_index,spec->map_input_index,index_size,hipMemcpyHostToDevice);
	spec->map_block_num=((spec->map_input_num)+(spec->map_thread_num)-1)/(spec->map_thread_num);
//	printf("%d\n",spec->map_block_num);
	map_count_warp<<<spec->map_block_num,spec->map_thread_num>>>(d_map_input_keys,d_map_input_values,d_map_input_index,d_map_im_key_size,d_map_im_value_size,d_map_im_num,spec->map_input_num);
	hipMemcpy(spec->map_im_key_size,d_map_im_key_size,map_im_size,hipMemcpyDeviceToHost);
	hipMemcpy(spec->map_im_value_size,d_map_im_value_size,map_im_size,hipMemcpyDeviceToHost);
	hipMemcpy(spec->map_im_num,d_map_im_num,map_im_size,hipMemcpyDeviceToHost);
//	printf("%s\n",spec->map_input_values);
//	printf("%d %d %d\n",*(spec->map_im_key_size),*(spec->map_im_value_size),*(spec->map_im_num));
	// cudaFree(d_map_input_keys);
	// cudaFree(d_map_input_values);
	// cudaFree(d_map_input_index);
	hipFree(d_map_im_key_size);
	hipFree(d_map_im_value_size);
	hipFree(d_map_im_num);
}

__device__ void im_emit(char* key,int start, int end, int value,char* im_key,int* im_value,Index* im_index,int g_im_key_offset,int g_im_value_offset,int key_local_offset,int word_num){
	int i;
//	static __shared__ int j=0;
	// for(i=start;i<end;i++){
	// 	*(im_key+key_local_offset+i-start)=*(key+i);
	// 	//*(im_key+(*j))='d';
	// 	//(*j)++;
	// }
	*im_key='i';*(im_key+1)='p';
//	*im_key='t';*(im_key+1)='s';
	*(im_value+word_num)=value;
	(im_index+word_num)->key_offset=g_im_key_offset+start;
	(im_index+word_num)->key_size=end-start;
	(im_index+word_num)->value_offset=g_im_value_offset+word_num;
	(im_index+word_num)->value_size=1;
}

__device__ void map(char*key,char*value,size_t key_size, size_t value_size,char* im_key,int*im_value,int im_key_size,int im_value_size,Index* im_index,int g_im_key_offset,int g_im_value_offset,int g_im_index_offset){
	int i=0;
	int start;
	int local_offset=0;
	int key_local_offset=0;
	int word_num=0;
	while(i<value_size){
		while((i<value_size)&&!isChar(*(value+i)))
			i++;
		start = i;
		while((i<value_size)&&isChar(*(value+i)))
			i++;
		if(start<i){
			if(key_local_offset==0){
				im_emit(value,start,i,1,im_key,im_value,im_index+g_im_index_offset,g_im_key_offset,g_im_value_offset,key_local_offset,word_num);}

			//break;
		}
		key_local_offset=key_local_offset+(i-start);
		//key_local_offset++;
		word_num++;
	}
}

__global__ void map_warp(char*input_keys,char*input_values,Index*input_index,char*im_key,int*im_value,Index*im_index,Index*im_loc,int* im_index_loc,int input_num){
	int i=blockDim.x*blockIdx.x+threadIdx.x;
	if(i==0){
		map((input_keys+((input_index+i)->key_offset)),(input_values+((input_index+i)->value_offset)),(input_index+i)->key_size,(input_index+i)->value_size,(im_key+(im_loc+i)->key_offset),(im_value+(im_loc+i)->value_offset),(im_loc+i)->key_size,(im_loc+i)->value_size,im_index,(im_loc+i)->key_offset,(im_loc+i)->value_offset,*(im_index_loc+i));
	}
}

void map_phase(MapReduceSpec* spec){
	int im_key_total_size=0;
	int im_value_total_size=0;
	int im_num_total=0;
	Index im_loc[spec->map_input_num];
	int im_index_loc[spec->map_input_num];

	for(int i=0; i<spec->map_input_num;i++){
		im_loc[i].key_offset=im_key_total_size;
		im_loc[i].key_size=*(spec->map_im_key_size+i);
		im_loc[i].value_offset=im_value_total_size>>2;
		im_loc[i].value_size=1;
		im_index_loc[i]=im_num_total;
		im_key_total_size+=*(spec->map_im_key_size+i);
		im_value_total_size+=*(spec->map_im_key_size+i);
		im_num_total+=*(spec->map_im_num+i);
	}
	printf("Map outpu entries: %d\n",im_num_total);	

	spec->im_keys=(char*)malloc(im_key_total_size);
	spec->im_values=(int*)malloc(im_value_total_size);
	spec->im_index=(Index*)malloc(im_num_total*sizeof(Index));
	
	char *d_im_keys;
	int *d_im_values;
	Index *d_im_index;
	Index *d_im_loc;
	int *d_im_index_loc;
	hipMalloc(&d_im_keys,im_key_total_size);
	hipMalloc(&d_im_values,im_value_total_size);
	hipMalloc(&d_im_index,im_num_total*sizeof(Index));
	hipMalloc(&d_im_loc,spec->map_input_num*sizeof(Index));
	hipMemcpy(d_im_loc,im_loc,spec->map_input_num*sizeof(Index),hipMemcpyHostToDevice);
	hipMalloc(&d_im_index_loc,spec->map_input_num*sizeof(int));
	hipMemcpy(d_im_index_loc,im_index_loc,spec->map_input_num*sizeof(int),hipMemcpyHostToDevice);
	map_warp<<<spec->map_block_num,spec->map_thread_num>>>(d_map_input_keys,d_map_input_values,d_map_input_index,d_im_keys,d_im_values,d_im_index,d_im_loc,d_im_index_loc,spec->map_input_num);
	hipMemcpy(spec->im_keys,d_im_keys,im_key_total_size,hipMemcpyDeviceToHost);
	hipMemcpy(spec->im_values,d_im_values,im_value_total_size,hipMemcpyDeviceToHost);
	hipMemcpy(spec->im_index,d_im_index,im_num_total*sizeof(Index),hipMemcpyDeviceToHost);	
	printf("%s\n%s\n",spec->im_keys,spec->map_input_values);
	
	free(spec->map_input_keys);
	free(spec->map_input_values);
	free(spec->map_input_index);
	free(spec->map_im_key_size);
	hipFree(d_map_input_keys);
	hipFree(d_map_input_values);
	hipFree(d_map_input_index);
	hipFree(d_im_keys);
	hipFree(d_im_values);
	hipFree(d_im_index);
	hipFree(d_im_loc);
	hipFree(d_im_index_loc);
}

void add_input_path(char *path,MapReduceSpec* spec){
	MapFileList* plist;
	plist=(MapFileList*)malloc(sizeof(MapFileList));
	spec->map_file_list=plist;
	struct dirent* entry = NULL;
	DIR *pDir;
	pDir=opendir(path);
	while((entry=readdir(pDir))!=NULL){
		if(entry->d_type==DT_REG){
			plist->filename=(char*)malloc(strlen(path)+strlen(entry->d_name)+1);
			strcpy(plist->filename,path);
		       	strcat(plist->filename,entry->d_name);
			plist->next=(MapFileList*)malloc(sizeof(MapFileList));
			plist=plist->next;
		}
	}
	map_input_split(spec);
	map_count_phase(spec);
	map_phase(spec);
}

MAP_COUNT{
	unsigned int i;
	unsigned int im_key_size=0;
	unsigned int im_value_size=0;
	int word_num=0;
	for(i=0;i<value_size;){
		while((i<value_size)&&!isChar(*(value+i)))
			i++;
		int start = i;
		while((i<value_size)&&isChar(*(value+i)))
			i++;
		if(start<i){
			im_key_size+=(i-start);
			im_value_size+=sizeof(int);
			word_num++;
		}
	}
	EMIT_IM_COUNT(im_key_size,im_value_size);
	//emitMapCount(im_key_size,im_value_size,word_num,key_im_size,value_im_size,map_im_num,threadID);
}

int main(int argc, char **argv){
	MapReduceSpec* spec=(MapReduceSpec*)malloc(sizeof(MapReduceSpec));
	init_mapreduce_spec(spec);
	add_input_path(argv[1],spec);
	free(spec);
}
